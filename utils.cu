#include "hip/hip_runtime.h"
#include "utils.h"
#include "debug.h"
#include "messages.h"

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
static const char* cublasGetErrorEnum(hipblasStatus_t error){
    switch (error){
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      throw_error("CUDA Error: " << hipGetErrorString(stat) << " at " << file << ":" << line);
      exit(1);
   }
}
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      throw_error("cuBlas Error: " << cublasGetErrorEnum(stat) << " at " << file << ":" << line);
      exit(1);
   }
}
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      throw_error("cuRand Error: " << stat << " at " << file << ":" << line);
      exit(1);
   }
}
void randn(float *array, int size) {
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
    hiprandGenerateUniform(prng, array, size);
    hiprandDestroyGenerator(prng);
}
bool has_nan(const float* a, size_t size){
    float* res;
    cudaErrCheck(hipMalloc((void**) &res, sizeof(float)*size));
    thrust::device_ptr<const float> p_a(a);
    thrust::device_ptr<float> p_res(res);
    thrust::transform(thrust::device, p_a, p_a+size, p_res, NaNTest());
    bool result = thrust::reduce(thrust::device, p_res, p_res+size);
    cudaErrCheck(hipFree(res));
    return result;
}
