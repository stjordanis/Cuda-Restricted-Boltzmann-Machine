#include "hip/hip_runtime.h"
#include "utils.h"

#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s at %s %d\n", hipGetErrorString(stat), file, line);
      exit(1);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d at %s %d\n", stat, file, line);
      exit(1);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d at %s %d\n", stat, file, line);
      exit(1);
   }
}

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }

__forceinline__ __device__ float sigmoidf(float in) {
    // raw approximation to sigmoid function
    return in / (1.f + fabsf(in));  
}

__global__
struct do_sample{
    __host__ __device__
    int operator()(const float n) const{
        return n > 0.5;  
    }
};
__global__
struct sigmoid{
    __host__ __device__
    float operator()(const float n) const{
        return n / (1.f + abs(n));  
    }
};
hipblasHandle_t& cublas_handle(){
    static hipblasHandle_t handle = NULL;
    if(handle == NULL){
        hipblasStatus_t stat;
        stat = hipblasCreate(&handle);
        if(stat != HIPBLAS_STATUS_SUCCESS)
            errx(1, "CUBLAS initialization failed\n");
    }
    return handle;
}
void randn(float *array, int size) {
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
    hiprandGenerateUniform(prng, array, size);
    hiprandDestroyGenerator(prng);
}
// z(m,n) = x(m,k) * y(k,n)
void matrix_mul(const float* x, const float*y, float* z, int xi, int xj, int yi, int yj, int zj){
    float alpha = 1.0, beta = 0.0;
    int m = yj, n = xi, k = yi;
    cublasErrCheck(hipblasSgemm(
        cublas_handle(),
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        m, n, k,
        &alpha,
        y, yj,
        x, xj,
        &beta,
        z, zj
    ));
}
void matrix_mul_tranpose_first(const float* x, const float*y, float* z, int xi, int xj, int yi, int yj, int zj){
    float alpha = 1.0, beta = 0.0;
    int m = yj, n = xj, k = yi;
    cublasErrCheck(hipblasSgemm(
        cublas_handle(),
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        m, n, k,
        &alpha,
        y, yj,
        x, xj,
        &beta,
        z, zj
    ));
}
void matrix_mul_tranpose_second(const float* x, const float*y, float* z, int xi, int xj, int yi, int yj, int zj){
    float alpha = 1.0, beta = 0.0;
    int m = yi, n = xi, k = yj;
    cublasErrCheck(hipblasSgemm(
        cublas_handle(),
        HIPBLAS_OP_T,
        HIPBLAS_OP_N,
        m, n, k,
        &alpha,
        y, yj,
        x, xj,
        &beta,
        z, zj
    ));
}
// x = sigmoid(x + y)
__global__ void add_sigmoid(float* x, const float* y, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i < size )
        x[i] += sigmoidf(x[i] + y[i]);
}
__global__ void add_diff(float* a, const float* x, const float* y, const float c, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i < size )
        a[i] += c*(x[i] - y[i]);
}
// a = a + outer(x,y)
void add_outer_prod(float* a, const float* x, const float* y, int nrow, int ncol, float alpha){
    hipblasStatus_t stat;
    stat = hipblasSger(
        cublas_handle(),
        ncol, nrow,
        &alpha,
        y, 1,
        x, 1,
        a, ncol
    );
    if(stat != HIPBLAS_STATUS_SUCCESS)
        errx(1,"CUBLAS outer prodduct error\n");
}
