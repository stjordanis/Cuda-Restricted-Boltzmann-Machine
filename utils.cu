#include "hip/hip_runtime.h"
#include "utils.h"

#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s at %s %d\n", hipGetErrorString(stat), file, line);
      exit(1);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d at %s %d\n", stat, file, line);
      exit(1);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d at %s %d\n", stat, file, line);
      exit(1);
   }
}

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }

__forceinline__ __device__ float sigmoidf(float in) {
    return in / (1.f + fabsf(in));  
}
__global__ void vectorAdd(float *y, float *a,  float *b, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a[i] + b[i];
}
__global__ void sigmoid(float *y, float *a, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = sigmoidf(a[i]);
}
__global__
struct do_sample{
    __host__ __device__
    int operator()(const float n) const{
        return n > 0.5;  
    }
};
__global__
struct sigmoid{
    __host__ __device__
    float operator()(const float n) const{
        return n / (1.f + abs(n));  
    }
};
hipblasHandle_t& cublasHandle(){
    static hipblasHandle_t handle = NULL;
    if(handle == NULL){
        hipblasStatus_t stat;
        stat = hipblasCreate(&handle);
        if(stat != HIPBLAS_STATUS_SUCCESS)
            errx(1, "CUBLAS initialization failed\n");
    }
    return handle;
}
void randn(float *array, int size) {
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
    hiprandGenerateUniform(prng, array, size);
    hiprandDestroyGenerator(prng);
}
// z(m,n) = x(m,k) * y(k,n)
void matrixMul(const float* x, const float*y, float* z, int xi, int xj, int yi, int yj, int zj){
    float alpha = 1.0, beta = 0.0;
    int m = yj, n = xi, k = yi;
    cublasErrCheck(hipblasSgemm(
        cublasHandle(),
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        m, n, k,
        &alpha,
        y, yj,
        x, xj,
        &beta,
        z, zj
    ));
}
void matrixMulTranposeFirst(const float* x, const float*y, float* z, int xi, int xj, int yi, int yj, int zj){
    float alpha = 1.0, beta = 0.0;
    int m = yj, n = xj, k = yi;
    cublasErrCheck(hipblasSgemm(
        cublasHandle(),
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        m, n, k,
        &alpha,
        y, yj,
        x, xj,
        &beta,
        z, zj
    ));
}
void matrixMulTranposeSecond(const float* x, const float*y, float* z, int xi, int xj, int yi, int yj, int zj){
    float alpha = 1.0, beta = 0.0;
    int m = yi, n = xi, k = yj;
    cublasErrCheck(hipblasSgemm(
        cublasHandle(),
        HIPBLAS_OP_T,
        HIPBLAS_OP_N,
        m, n, k,
        &alpha,
        y, yj,
        x, xj,
        &beta,
        z, zj
    ));
}
// x = sigmoid(x + y)
__global__ void add_sigmoid(float* x, const float* y, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i < size )
        x[i] += sigmoidf(x[i] + y[i]);
}
__global__ void add_diff(float* a, const float* x, const float* y, const float c, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i < size )
        a[i] += c*(x[i] - y[i]);
}
// a = a + outer(x,y)
void add_outer_prod(float* a, const float* x, const float* y, int nrow, int ncol, float alpha){
    hipblasStatus_t stat;
    stat = hipblasSger(
        cublasHandle(),
        ncol, nrow,
        &alpha,
        y, 1,
        x, 1,
        a, ncol
    );
    if(stat != HIPBLAS_STATUS_SUCCESS)
        errx(1,"CUBLAS outer prodduct error\n");
}
