#include "hip/hip_runtime.h"
#include "utils.h"
#include "debug.h"
#include "messages.h"

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
static const char* cublasGetErrorEnum(hipblasStatus_t error){
    switch (error){
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      throw_error("CUDA Error: " << hipGetErrorString(stat) << " at " << file << ":" << line);
      exit(1);
   }
}
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      throw_error("cuBlas Error: " << cublasGetErrorEnum(stat) << " at " << file << ":" << line);
      exit(1);
   }
}
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      throw_error("cuRand Error: " << stat << " at " << file << ":" << line);
      exit(1);
   }
}

hipblasHandle_t& cublas_handle(){
    static hipblasHandle_t handle = NULL;
    if(handle == NULL)
        cublasErrCheck(hipblasCreate(&handle));
    return handle;
}
// z(m,n) = x(m,k) * y(k,n)
void matrix_mul(const float* x, const float*y, float* z, int xi, int xj, int yi, int yj, int zj){
    float alpha = 1.0, beta = 0.0;
    int m = yj, n = xi, k = yi;
    cublasErrCheck(hipblasSgemm(
        cublas_handle(),
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        m, n, k,
        &alpha,
        y, yj,
        x, xj,
        &beta,
        z, zj
    ));
}
void matrix_mul_tranpose_first(const float* x, const float*y, float* z, int xi, int xj, int yi, int yj, int zj){
    float alpha = 1.0, beta = 0.0;
    int m = yj, n = xj, k = yi;
    cublasErrCheck(hipblasSgemm(
        cublas_handle(),
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        m, n, k,
        &alpha,
        y, yj,
        x, xj,
        &beta,
        z, zj
    ));
}
void matrix_mul_tranpose_second(const float* x, const float*y, float* z, int xi, int xj, int yi, int yj, int zj){
    float alpha = 1.0, beta = 0.0;
    int m = yi, n = xi, k = yj;
    cublasErrCheck(hipblasSgemm(
        cublas_handle(),
        HIPBLAS_OP_T,
        HIPBLAS_OP_N,
        m, n, k,
        &alpha,
        y, yj,
        x, xj,
        &beta,
        z, zj
    ));
}
// a = a + outer(x,y)
void add_outer_prod(float* a, const float* x, const float* y, int nrow, int ncol, float alpha){
    hipblasStatus_t stat;
    stat = hipblasSger(
        cublas_handle(),
        ncol, nrow,
        &alpha,
        y, 1,
        x, 1,
        a, ncol
    );
    if(stat != HIPBLAS_STATUS_SUCCESS){
        throw_error("CUBLAS outer prodduct error");
        exit(1);
    }
}

void randn(float *array, int size) {
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
    hiprandGenerateUniform(prng, array, size);
    hiprandDestroyGenerator(prng);
}

__global__ void setup_random_numbers(hiprandState * state, unsigned long seed){
    int id = threadIdx.x;
    hiprand_init( seed, id, 0, &state[id] );
} 

__global__ void transform_example_kernel(float* to, char* from, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i < size ){
        to[i] = ((float)(from[i])) * 2.0f / 255.0f - 1.0f;
    }
}

void transform_example(float* gpu_buffer, char* gpu_tmp, char* cpu_buffer, int size){
    const int bsize = 128;
    const int gsize = CeilDiv(size,bsize);
    cudaErrCheck(hipMemcpy(gpu_tmp, cpu_buffer, size, hipMemcpyHostToDevice));
    transform_example_kernel<<<bsize, gsize>>>(gpu_buffer, gpu_tmp, size);
}
